#include "hip/hip_runtime.h"
﻿#include "bgmodel.h"
#include<iostream>
#include<cv.h>
#include<cstdio>
#include<highgui.h>
#include<windows.h>  
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"

using namespace cv;
using namespace std;


__global__ void Update(unsigned char* inputImg, unsigned char* outputImg, float* Probability)
{

	int idx = blockIdx.x * (BLOCKS_WID * BLOCKS_HEI) * 256 + (blockIdx.z * BLOCKS_WID + blockIdx.y) * 256 + threadIdx.y * blockDim.x + threadIdx.x;
	//n*(40*30) * 256 + (blk_y * 40 + blk_x) * 256 + threadIdx.y * 16 + threadIdx.x;
	int binIDX = idx * BIN_NUM;
	int currentBin = inputImg[idx] / (DELTA_B);
	if (Probability[binIDX + currentBin] < THRESHOLD) outputImg[idx / 3] = 255;
	for (int bin = 0; bin < BIN_NUM; bin++) {
		Probability[binIDX + bin] *= (BETA);
	}
	Probability[binIDX + currentBin] += ALPHA;
}

__global__ void Init(unsigned char* inputImg, float* Probability)
{
	int idx = blockIdx.x * (BLOCKS_WID * BLOCKS_HEI) * 256 + (blockIdx.z * BLOCKS_WID + blockIdx.y) * 256 + threadIdx.y * blockDim.x + threadIdx.x;
	int binIDX = idx * BIN_NUM;
	int currentBin = inputImg[idx] / (DELTA_B);
	Probability[binIDX + currentBin] = 1.0f;
}

int main(void)
{
	dim3 dimBlock(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
	dim3 dimGrid(3, BLOCKS_WID, BLOCKS_HEI);

	IplImage* InputImg;
	IplImage* resizeImg;
	IplImage* OutputImg;

	hipSetDevice(1);

	OutputImg = cvCreateImage(cvSize(PIX_WID, PIX_HEI), 8, 1);
	resizeImg = cvCreateImage(cvSize(PIX_WID, PIX_HEI), 8, 3);
	unsigned char* imgBuffer;
	unsigned char* outBuffer;

	unsigned char* cudaInputBuffer;
	unsigned char* cudaOutputBuffer;
	float* cudaProbability;
	outBuffer = (unsigned char*)OutputImg->imageData;
	hipMalloc((void**)&cudaInputBuffer, TOTAL_PIX * 3);
	hipMalloc((void**)&cudaOutputBuffer, TOTAL_PIX);
	hipMalloc((void**)&cudaProbability, BIN_NUM * TOTAL_PIX * 3 * sizeof(float));
	hipMemset(cudaProbability, 0, BIN_NUM * TOTAL_PIX * 3 * sizeof(float));
	char fileName[128] = { '\0' };
	int count = 2;

	sprintf(fileName, "D:\\CoVision\\PETS2006\\input\\in000001.jpg");
	InputImg = cvLoadImage(fileName);
	cvCvtColor(InputImg, InputImg, CV_BGR2YCrCb);
	cvResize(InputImg, resizeImg);
	imgBuffer = (unsigned char*)resizeImg->imageData;

	hipMemcpy(cudaInputBuffer, imgBuffer, TOTAL_PIX * 3, hipMemcpyHostToDevice);
	Init << <dimGrid, dimBlock >> > (cudaInputBuffer, cudaProbability);

	float totalTime = 0;

	while (count < 1200) {

		if (InputImg != NULL) cvReleaseImage(&InputImg);
		sprintf(fileName, "D:\\CoVision\\PETS2006\\input\\in%06d.jpg", count);
		count++;

		InputImg = cvLoadImage(fileName);
		cvCvtColor(InputImg, InputImg, CV_BGR2YCrCb);
		cvResize(InputImg, resizeImg);

		hipEvent_t start, stop;
		float time;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);

		imgBuffer = (unsigned char*)resizeImg->imageData;
		hipMemcpy(cudaInputBuffer, imgBuffer, TOTAL_PIX * 3, hipMemcpyHostToDevice);
		hipMemset(cudaOutputBuffer, 0, TOTAL_PIX);

		Update << <dimGrid, dimBlock >> > (cudaInputBuffer, cudaOutputBuffer, cudaProbability);
		hipDeviceSynchronize();
		hipMemcpy(outBuffer, cudaOutputBuffer, TOTAL_PIX, hipMemcpyDeviceToHost);

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time, start, stop);
		hipEventDestroy(start);
		hipEventDestroy(stop);
		totalTime += time;

		//cvShowImage("Input", resizeImg);
		//cvShowImage("Foreground", OutputImg);
		//cvWaitKey(1);
	}

	cout << "fps: " << 1200 / (totalTime / 1000) << endl;
	cout << totalTime << endl;

	hipFree(cudaInputBuffer);
	hipFree(cudaOutputBuffer);
	hipFree(cudaProbability);
	cudaInputBuffer = NULL;
	cudaOutputBuffer = NULL;
	cudaProbability = NULL;
	imgBuffer = NULL;

	delete[] cudaProbability;
	if (InputImg != NULL) cvReleaseImage(&InputImg);
	if (OutputImg != NULL) cvReleaseImage(&OutputImg);
	if (resizeImg != NULL) cvReleaseImage(&resizeImg);
	InputImg = NULL;
	OutputImg = NULL;
	resizeImg = NULL;
	return 0;
}